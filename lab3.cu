#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }

__global__ void turnmat(uchar *image, uchar *out_image, int rows, int cols)
{
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= rows || j >= cols)
        return;

    uchar *p = image + 3 * (i * cols + j);
    int out_i = cols - 1 - j;
    int out_j = i;
    uchar *out_p = out_image + 3 * (out_i * rows + out_j);

    for (int ch = 0; ch < 3; ch++)
      *(out_p + ch) = *(p + ch);
}

// __global__ void shared_turnmat(uchar *image, uchar *out_image, int rows, int cols)
// {
//     //__shared__ uchar* temp[BLOCK_DIM][BLOCK_DIM]; // uchar
//     __shared__ uchar* temp;//[blockDim.y][blockDim.x];
//     CHECK( hipMalloc(&temp, 3 * blockDim.x * blockDim.y) );
//
//     int i = threadIdx.y + blockIdx.y * blockDim.y;
//     int j = threadIdx.x + blockIdx.x * blockDim.x;
//     if (i >= rows || j >= cols)
//         return;
//
//     uchar3 *p = image + i * cols + j; // 3 *
//     int new_i = blockDim.y - ((j + 1) % blockDim.y);
//     int new_j = i % blockDim.x;
//     temp[new_i][new_j] = p[i][j];
//
//     for (int ch = 0; ch < 3; ch++)
//       *(temp + ch) = *(p + ch);
//
//     __syncthreads();
//
//     int out_i = cols - 1 - j;
//     int out_j = i;
//     out_image[out_i][out_j] = temp[new_i][new_j];
//
//     // int ty = threadIdx.y;
//     // int tx = threadIdx.x;
//     // int by = blockIdx.y;
//     // int bx = blockIdx.x;
//     // int i = ty + by * blockDim.y;
//     // int j = tx + bx * blockDim.x;
//     // if (i >= rows || j >= cols)
//     //     return;
// }

int main(void)
{
    int N = 10*1000*1000;
    Mat image;

    image = imread("pic.jpeg", CV_LOAD_IMAGE_COLOR);   // Read the file
    if(! image.data )                              // Check for invalid input
    {
        cout << "Could not open or find the image" << std::endl ;
        return -1;
    }

    //Mat out_image1(image.cols, image.rows, DataType<Vec3b>::type);

    hipEvent_t startCUDA, stopCUDA;
    //clock_t startCPU;
    float elapsedTimeCUDA/*, elapsedTimeCPU*/;
    hipEventCreate(&startCUDA);
    hipEventCreate(&stopCUDA);

    // startCPU = clock();
    //
    // for (int i = 0; i < image.rows; i++)
    // {
    //     Vec3b* p = image.ptr<Vec3b>(i);
    //     for (int j = 0; j < image.cols; j++)
    //     {
    //         Vec3b* out_p = out_image1.ptr<Vec3b>(j);
    //         out_p[i] = p[image.cols - j - 1];
    //     }
    // }
    //
    // elapsedTimeCPU = (double)(clock()-startCPU)/CLOCKS_PER_SEC;
    // cout << "CPU sum time = " << elapsedTimeCPU*1000 << " ms\n";
    // cout << "CPU memory throughput = " << 3*N*sizeof(float)/elapsedTimeCPU/1024/1024/1024 << " Gb/s\n";
    //
    // imwrite("pic_resCPU.jpeg", out_image1);

////////////////////////////////////////////////////////////////////////////////////////////////////

    Mat out_image(image.cols, image.rows, DataType<Vec3b>::type);
    uchar *dev_src_image;

    uchar * res_src_image;
    CHECK( hipMalloc(&res_src_image, 3 * out_image.cols * out_image.rows) );
    CHECK( hipMemcpy(res_src_image, out_image.data, 3 * out_image.cols * out_image.rows, hipMemcpyHostToDevice) );

    CHECK( hipMalloc(&dev_src_image, 3 * image.cols * image.rows) );
    CHECK( hipMemcpy(dev_src_image, image.data, 3 * image.cols * image.rows, hipMemcpyHostToDevice) );

    hipEventRecord(startCUDA,0);

    turnmat<<<dim3((image.cols + 15) / 16, (image.rows + 15) / 16, 1), dim3(16, 16, 1)>>>(dev_src_image, res_src_image, image.rows, image.cols);

    hipEventRecord(stopCUDA,0);
    hipEventSynchronize(stopCUDA);
    CHECK(hipGetLastError());

    hipEventElapsedTime(&elapsedTimeCUDA, startCUDA, stopCUDA);

    cout << "CUDA sum time = " << elapsedTimeCUDA << " ms\n";
    cout << "CUDA memory throughput = " << 3*N*sizeof(float)/elapsedTimeCUDA/1024/1024/1.024 << " Gb/s\n";
    CHECK(hipMemcpy(out_image.data, res_src_image, 3 * image.cols * image.rows, hipMemcpyDeviceToHost));

    imwrite("pic_resGPU.jpeg", out_image);
    return 0;

////////////////////////////////////////////////////////////////////////////////////////////////////
}
